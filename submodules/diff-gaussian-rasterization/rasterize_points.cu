/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include <math.h>
#include <torch/extension.h>
#include <cstdio>
#include <sstream>
#include <iostream>
#include <tuple>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <memory>
#include "cuda_rasterizer/config.h"
#include "cuda_rasterizer/rasterizer.h"
#include <fstream>
#include <string>
#include <functional>

std::function<char*(size_t N)> resizeFunctional(torch::Tensor& t) {
    auto lambda = [&t](size_t N) {
        t.resize_({(long long)N});
		return reinterpret_cast<char*>(t.contiguous().data_ptr());
    };
    return lambda;
}

std::tuple<int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
RasterizeGaussiansCUDA(
	const torch::Tensor& background,
	const torch::Tensor& means3D,
    const torch::Tensor& colors,
    const torch::Tensor& opacity,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
	const torch::Tensor& projmatrix,
	const float tan_fovx, 
	const float tan_fovy,
    const int image_height,
    const int image_width,
	const torch::Tensor& sh,
	const int degree,
	const torch::Tensor& campos,
	const bool prefiltered,
	const bool debug)
{
  if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
    AT_ERROR("means3D must have dimensions (num_points, 3)");
  }
  
  const int P = means3D.size(0);
  const int H = image_height;
  const int W = image_width;

  auto int_opts = means3D.options().dtype(torch::kInt32);
  auto float_opts = means3D.options().dtype(torch::kFloat32);

  torch::Tensor out_color = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);
  torch::Tensor radii = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
  
  torch::Device device(torch::kCUDA);
  torch::TensorOptions options(torch::kByte);
  torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
  torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
  torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
  std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
  std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
  std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);
  
  int rendered = 0;
  if(P != 0)
  {
	  int M = 0;
	  if(sh.size(0) != 0)
	  {
		M = sh.size(1);
      }

	  rendered = CudaRasterizer::Rasterizer::forward(
	    geomFunc,
		binningFunc,
		imgFunc,
	    P, degree, M,
		background.contiguous().data<float>(),
		W, H,
		means3D.contiguous().data<float>(),
		sh.contiguous().data_ptr<float>(),
		colors.contiguous().data<float>(), 
		opacity.contiguous().data<float>(), 
		scales.contiguous().data_ptr<float>(),
		scale_modifier,
		rotations.contiguous().data_ptr<float>(),
		cov3D_precomp.contiguous().data<float>(), 
		viewmatrix.contiguous().data<float>(), 
		projmatrix.contiguous().data<float>(),
		campos.contiguous().data<float>(),
		tan_fovx,
		tan_fovy,
		prefiltered,
		out_color.contiguous().data<float>(),
		radii.contiguous().data<int>(),
		debug);
  }
  return std::make_tuple(rendered, out_color, radii, geomBuffer, binningBuffer, imgBuffer);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
 RasterizeGaussiansBackwardCUDA(
 	const torch::Tensor& background,
	const torch::Tensor& means3D,
	const torch::Tensor& radii,
    const torch::Tensor& colors,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
    const torch::Tensor& projmatrix,
	const float tan_fovx,
	const float tan_fovy,
    const torch::Tensor& dL_dout_color,
	const torch::Tensor& sh,
	const int degree,
	const torch::Tensor& campos,
	const torch::Tensor& geomBuffer,
	const int R,
	const torch::Tensor& binningBuffer,
	const torch::Tensor& imageBuffer,
	const bool debug) 
{
  const int P = means3D.size(0);
  const int H = dL_dout_color.size(1);
  const int W = dL_dout_color.size(2);
  
  int M = 0;
  if(sh.size(0) != 0)
  {	
	M = sh.size(1);
  }

  torch::Tensor dL_dmeans3D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dmeans2D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dcolors = torch::zeros({P, NUM_CHANNELS}, means3D.options());
  torch::Tensor dL_dconic = torch::zeros({P, 2, 2}, means3D.options());
  torch::Tensor dL_dopacity = torch::zeros({P, 1}, means3D.options());
  torch::Tensor dL_dcov3D = torch::zeros({P, 6}, means3D.options());
  torch::Tensor dL_dsh = torch::zeros({P, M, 3}, means3D.options());
  torch::Tensor dL_dscales = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_drotations = torch::zeros({P, 4}, means3D.options());
  
  if(P != 0)
  {  
	  CudaRasterizer::Rasterizer::backward(P, degree, M, R,
	  background.contiguous().data<float>(),
	  W, H, 
	  means3D.contiguous().data<float>(),
	  sh.contiguous().data<float>(),
	  colors.contiguous().data<float>(),
	  scales.data_ptr<float>(),
	  scale_modifier,
	  rotations.data_ptr<float>(),
	  cov3D_precomp.contiguous().data<float>(),
	  viewmatrix.contiguous().data<float>(),
	  projmatrix.contiguous().data<float>(),
	  campos.contiguous().data<float>(),
	  tan_fovx,
	  tan_fovy,
	  radii.contiguous().data<int>(),
	  reinterpret_cast<char*>(geomBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(binningBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(imageBuffer.contiguous().data_ptr()),
	  dL_dout_color.contiguous().data<float>(),
	  dL_dmeans2D.contiguous().data<float>(),
	  dL_dconic.contiguous().data<float>(),  
	  dL_dopacity.contiguous().data<float>(),
	  dL_dcolors.contiguous().data<float>(),
	  dL_dmeans3D.contiguous().data<float>(),
	  dL_dcov3D.contiguous().data<float>(),
	  dL_dsh.contiguous().data<float>(),
	  dL_dscales.contiguous().data<float>(),
	  dL_drotations.contiguous().data<float>(),
	  debug);
  }

  return std::make_tuple(dL_dmeans2D, dL_dcolors, dL_dopacity, dL_dmeans3D, dL_dcov3D, dL_dsh, dL_dscales, dL_drotations);
}

torch::Tensor markVisible(
		torch::Tensor& means3D,
		torch::Tensor& viewmatrix,
		torch::Tensor& projmatrix)
{ 
  const int P = means3D.size(0);
  
  torch::Tensor present = torch::full({P}, false, means3D.options().dtype(at::kBool));
 
  if(P != 0)
  {
	CudaRasterizer::Rasterizer::markVisible(P,
		means3D.contiguous().data<float>(),
		viewmatrix.contiguous().data<float>(),
		projmatrix.contiguous().data<float>(),
		present.contiguous().data<bool>());
  }
  
  return present;
}

//CountGaussian_cuda split
std::tuple<int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
CountGaussiansCUDA(
	const torch::Tensor& background,
	const torch::Tensor& means3D,
    const torch::Tensor& colors,
    const torch::Tensor& opacity,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
	const torch::Tensor& projmatrix,
	const float tan_fovx, 
	const float tan_fovy,
    const int image_height,
    const int image_width,
	const torch::Tensor& sh,
	const int degree,
	const torch::Tensor& campos,
	const bool prefiltered,
	const bool debug)
{
  if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
    AT_ERROR("means3D must have dimensions (num_points, 3)");
  }
  
  const int P = means3D.size(0);
  const int H = image_height;
  const int W = image_width;

  auto int_opts = means3D.options().dtype(torch::kInt32);
  auto float_opts = means3D.options().dtype(torch::kFloat32);

  torch::Tensor out_color = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);
  torch::Tensor radii = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
  torch::Device device(torch::kCUDA);
  torch::TensorOptions options(torch::kByte);
  torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
  torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
  torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
  std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
  std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
  std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);

  //initilize  gaussians_count and important_score 
  torch::Tensor gaussians_count = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
  torch::Tensor important_score = torch::full({P}, 0.0, float_opts);

  int rendered = 0;
  if(P != 0)
  {
	  int M = 0;
	  if(sh.size(0) != 0)
	  {
		M = sh.size(1);
      }
	  // overloaded forward, added gaussians_count, important_score
	  rendered = CudaRasterizer::Rasterizer::forwardCount(
	    geomFunc,
		binningFunc,
		imgFunc,
	    P, degree, M,
		background.contiguous().data<float>(),
		W, H,
		means3D.contiguous().data<float>(),
		sh.contiguous().data_ptr<float>(),
		colors.contiguous().data<float>(), 
		opacity.contiguous().data<float>(), 
		scales.contiguous().data_ptr<float>(),
		scale_modifier,
		rotations.contiguous().data_ptr<float>(),
		cov3D_precomp.contiguous().data<float>(), 
		viewmatrix.contiguous().data<float>(), 
		projmatrix.contiguous().data<float>(),
		campos.contiguous().data<float>(),
		tan_fovx,
		tan_fovy,
		prefiltered,
		out_color.contiguous().data<float>(),
		gaussians_count.contiguous().data<int>(),
		important_score.contiguous().data<float>(),
		radii.contiguous().data<int>(),
		debug);
  }
  return std::make_tuple(rendered, out_color, radii, geomBuffer, binningBuffer, imgBuffer, gaussians_count, important_score);
}

std::tuple<int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
BlendingWeightGaussiansCUDA(
	const torch::Tensor& background,
	const torch::Tensor& means3D,
    const torch::Tensor& colors,
    const torch::Tensor& opacity,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
	const torch::Tensor& projmatrix,
	const float tan_fovx, 
	const float tan_fovy,
    const int image_height,
    const int image_width,
	const torch::Tensor& sh,
	const int degree,
	const torch::Tensor& campos,
	const bool prefiltered,
	const bool debug)
{
  if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
    AT_ERROR("means3D must have dimensions (num_points, 3)");
  }
  
  const int P = means3D.size(0);
  const int H = image_height;
  const int W = image_width;

  auto int_opts = means3D.options().dtype(torch::kInt32);
  auto float_opts = means3D.options().dtype(torch::kFloat32);

  torch::Tensor out_color = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);
  torch::Tensor radii = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
  torch::Device device(torch::kCUDA);
  torch::TensorOptions options(torch::kByte);
  torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
  torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
  torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
  std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
  std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
  std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);

  // blending_weight_score
  torch::Tensor gaussians_count = torch::full({P}, 0, int_opts);
  torch::Tensor accum_max_count = torch::full({P}, 0, int_opts);
  torch::Tensor blending_weight_score = torch::full({P}, 0.0, float_opts);

  int rendered = 0;
  if(P != 0)
  {
	  int M = 0;
	  if(sh.size(0) != 0)
	  {
		M = sh.size(1);
      }
	  // overloaded forward, blending_weight_score
	  rendered = CudaRasterizer::Rasterizer::forwardBWScore(
	    geomFunc,
		binningFunc,
		imgFunc,
	    P, degree, M,
		background.contiguous().data<float>(),
		W, H,
		means3D.contiguous().data<float>(),
		sh.contiguous().data_ptr<float>(),
		colors.contiguous().data<float>(), 
		opacity.contiguous().data<float>(), 
		scales.contiguous().data_ptr<float>(),
		scale_modifier,
		rotations.contiguous().data_ptr<float>(),
		cov3D_precomp.contiguous().data<float>(), 
		viewmatrix.contiguous().data<float>(), 
		projmatrix.contiguous().data<float>(),
		campos.contiguous().data<float>(),
		tan_fovx,
		tan_fovy,
		prefiltered,
		out_color.contiguous().data<float>(),
		gaussians_count.contiguous().data<int>(),
		accum_max_count.contiguous().data<int>(),
		blending_weight_score.contiguous().data<float>(),
		radii.contiguous().data<int>(),
		debug);
  }
  return std::make_tuple(rendered, out_color, radii, geomBuffer, binningBuffer, imgBuffer, gaussians_count, accum_max_count, blending_weight_score);
}

std::tuple<int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
MaxWeightGaussiansCUDA(
	const torch::Tensor& background,
	const torch::Tensor& means3D,
    const torch::Tensor& colors,
    const torch::Tensor& opacity,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
	const torch::Tensor& projmatrix,
	const float tan_fovx, 
	const float tan_fovy,
    const int image_height,
    const int image_width,
	const torch::Tensor& sh,
	const int degree,
	const torch::Tensor& campos,
	const bool prefiltered,
	const bool debug)
{
  if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
    AT_ERROR("means3D must have dimensions (num_points, 3)");
  }
  
  const int P = means3D.size(0);
  const int H = image_height;
  const int W = image_width;

  auto int_opts = means3D.options().dtype(torch::kInt32);
  auto float_opts = means3D.options().dtype(torch::kFloat32);

  torch::Tensor out_color = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);
  torch::Tensor radii = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
  torch::Device device(torch::kCUDA);
  torch::TensorOptions options(torch::kByte);
  torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
  torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
  torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
  std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
  std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
  std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);

  // max_weight_score
  torch::Tensor max_weight_score = torch::full({P}, 0.0, float_opts);

  int rendered = 0;
  if(P != 0)
  {
	  int M = 0;
	  if(sh.size(0) != 0)
	  {
		M = sh.size(1);
      }
	  // overloaded forward, max_weight_score
	  rendered = CudaRasterizer::Rasterizer::forwardMWScore(
	    geomFunc,
		binningFunc,
		imgFunc,
	    P, degree, M,
		background.contiguous().data<float>(),
		W, H,
		means3D.contiguous().data<float>(),
		sh.contiguous().data_ptr<float>(),
		colors.contiguous().data<float>(), 
		opacity.contiguous().data<float>(), 
		scales.contiguous().data_ptr<float>(),
		scale_modifier,
		rotations.contiguous().data_ptr<float>(),
		cov3D_precomp.contiguous().data<float>(), 
		viewmatrix.contiguous().data<float>(), 
		projmatrix.contiguous().data<float>(),
		campos.contiguous().data<float>(),
		tan_fovx,
		tan_fovy,
		prefiltered,
		out_color.contiguous().data<float>(),
		max_weight_score.contiguous().data<float>(),
		radii.contiguous().data<int>(),
		debug);
  }
  return std::make_tuple(rendered, out_color, radii, geomBuffer, binningBuffer, imgBuffer, max_weight_score);
}

std::tuple<int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
TopKGaussiansCUDA(
	const torch::Tensor& background,
	const torch::Tensor& means3D,
    const torch::Tensor& colors,
    const torch::Tensor& opacity,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
	const torch::Tensor& projmatrix,
	const float tan_fovx, 
	const float tan_fovy,
    const int image_height,
    const int image_width,
	const torch::Tensor& sh,
	const int degree,
	const torch::Tensor& campos,
	const bool prefiltered,
	const bool debug,
	const int topk,
	const int score_function,
	const float p_dist_activation_coef,
	const float c_dist_activation_coef)
{
  if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
    AT_ERROR("means3D must have dimensions (num_points, 3)");
  }
  
  const int P = means3D.size(0);
  const int H = image_height;
  const int W = image_width;

  auto int_opts = means3D.options().dtype(torch::kInt32);
  auto float_opts = means3D.options().dtype(torch::kFloat32);
  auto bool_opts = means3D.options().dtype(torch::kBool);

  torch::Tensor out_color = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);
  torch::Tensor radii = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
  torch::Device device(torch::kCUDA);
  torch::TensorOptions options(torch::kByte);
  torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
  torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
  torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
  std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
  std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
  std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);

  // topk_mask
  torch::Tensor topk_mask = torch::full({P}, false, bool_opts);

  int rendered = 0;
  if(P != 0)
  {
	  int M = 0;
	  if(sh.size(0) != 0)
	  {
		M = sh.size(1);
      }
	  // overloaded forward, topk_mask
	  rendered = CudaRasterizer::Rasterizer::forwardTopK(
	    geomFunc,
		binningFunc,
		imgFunc,
	    P, degree, M,
		background.contiguous().data<float>(),
		W, H,
		means3D.contiguous().data<float>(),
		sh.contiguous().data_ptr<float>(),
		colors.contiguous().data<float>(), 
		opacity.contiguous().data<float>(), 
		scales.contiguous().data_ptr<float>(),
		scale_modifier,
		rotations.contiguous().data_ptr<float>(),
		cov3D_precomp.contiguous().data<float>(), 
		viewmatrix.contiguous().data<float>(), 
		projmatrix.contiguous().data<float>(),
		campos.contiguous().data<float>(),
		tan_fovx,
		tan_fovy,
		prefiltered,
		topk,
		score_function,
		p_dist_activation_coef,
		c_dist_activation_coef,
		out_color.contiguous().data<float>(),
		topk_mask.contiguous().data<bool>(),
		radii.contiguous().data<int>(),
		debug);
  }
  return std::make_tuple(rendered, out_color, radii, geomBuffer, binningBuffer, imgBuffer, topk_mask);
}

std::tuple<int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
TopKColorGaussiansCUDA(
	const torch::Tensor& background,
	const torch::Tensor& means3D,
    const torch::Tensor& colors,
    const torch::Tensor& opacity,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
	const torch::Tensor& projmatrix,
	const float tan_fovx, 
	const float tan_fovy,
    const int image_height,
    const int image_width,
	const torch::Tensor& sh,
	const int degree,
	const torch::Tensor& campos,
	const bool prefiltered,
	const bool debug,
	const int topk_color,
	const int score_function,
	const torch::Tensor& image_gt,
	const float p_dist_activation_coef,
	const float c_dist_activation_coef)
{
  if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
    AT_ERROR("means3D must have dimensions (num_points, 3)");
  }
  
  const int P = means3D.size(0);
  const int H = image_height;
  const int W = image_width;

  auto int_opts = means3D.options().dtype(torch::kInt32);
  auto float_opts = means3D.options().dtype(torch::kFloat32);
  auto bool_opts = means3D.options().dtype(torch::kBool);

  torch::Tensor out_color = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);
  torch::Tensor radii = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
  torch::Device device(torch::kCUDA);
  torch::TensorOptions options(torch::kByte);
  torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
  torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
  torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
  std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
  std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
  std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);

  // topk_color_mask
  torch::Tensor topk_color_mask = torch::full({P}, false, bool_opts);

  int rendered = 0;
  if(P != 0)
  {
	  int M = 0;
	  if(sh.size(0) != 0)
	  {
		M = sh.size(1);
      }
	  // overloaded forward, topk_color_mask
	  rendered = CudaRasterizer::Rasterizer::forwardTopKColor(
	    geomFunc,
		binningFunc,
		imgFunc,
	    P, degree, M,
		background.contiguous().data<float>(),
		W, H,
		means3D.contiguous().data<float>(),
		sh.contiguous().data_ptr<float>(),
		colors.contiguous().data<float>(), 
		opacity.contiguous().data<float>(), 
		scales.contiguous().data_ptr<float>(),
		scale_modifier,
		rotations.contiguous().data_ptr<float>(),
		cov3D_precomp.contiguous().data<float>(), 
		viewmatrix.contiguous().data<float>(), 
		projmatrix.contiguous().data<float>(),
		campos.contiguous().data<float>(),
		tan_fovx,
		tan_fovy,
		prefiltered,
		topk_color,
		score_function,
		image_gt.contiguous().data_ptr<float>(),
		p_dist_activation_coef,
		c_dist_activation_coef,
		out_color.contiguous().data<float>(),
		topk_color_mask.contiguous().data<bool>(),
		radii.contiguous().data<int>(),
		debug);
  }
  return std::make_tuple(rendered, out_color, radii, geomBuffer, binningBuffer, imgBuffer, topk_color_mask);
}

std::tuple<int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
TopKWeightGaussiansCUDA(
	const torch::Tensor& background,
	const torch::Tensor& means3D,
    const torch::Tensor& colors,
    const torch::Tensor& opacity,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
	const torch::Tensor& projmatrix,
	const float tan_fovx, 
	const float tan_fovy,
    const int image_height,
    const int image_width,
	const torch::Tensor& sh,
	const int degree,
	const torch::Tensor& campos,
	const bool prefiltered,
	const bool debug,
	const int topk_weight)
{
  if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
    AT_ERROR("means3D must have dimensions (num_points, 3)");
  }
  
  const int P = means3D.size(0);
  const int H = image_height;
  const int W = image_width;

  auto int_opts = means3D.options().dtype(torch::kInt32);
  auto float_opts = means3D.options().dtype(torch::kFloat32);
  auto bool_opts = means3D.options().dtype(torch::kBool);

  torch::Tensor out_color = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);
  torch::Tensor radii = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
  torch::Device device(torch::kCUDA);
  torch::TensorOptions options(torch::kByte);
  torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
  torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
  torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
  std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
  std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
  std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);

  // topk_weight_mask
  torch::Tensor topk_weight_mask = torch::full({P}, false, bool_opts);

  int rendered = 0;
  if(P != 0)
  {
	  int M = 0;
	  if(sh.size(0) != 0)
	  {
		M = sh.size(1);
      }
	  // overloaded forward, topk_weight_mask
	  rendered = CudaRasterizer::Rasterizer::forwardTopKWeight(
	    geomFunc,
		binningFunc,
		imgFunc,
	    P, degree, M,
		background.contiguous().data<float>(),
		W, H,
		means3D.contiguous().data<float>(),
		sh.contiguous().data_ptr<float>(),
		colors.contiguous().data<float>(), 
		opacity.contiguous().data<float>(), 
		scales.contiguous().data_ptr<float>(),
		scale_modifier,
		rotations.contiguous().data_ptr<float>(),
		cov3D_precomp.contiguous().data<float>(), 
		viewmatrix.contiguous().data<float>(), 
		projmatrix.contiguous().data<float>(),
		campos.contiguous().data<float>(),
		tan_fovx,
		tan_fovy,
		prefiltered,
		topk_weight,
		out_color.contiguous().data<float>(),
		topk_weight_mask.contiguous().data<bool>(),
		radii.contiguous().data<int>(),
		debug);
  }
  return std::make_tuple(rendered, out_color, radii, geomBuffer, binningBuffer, imgBuffer, topk_weight_mask);
}